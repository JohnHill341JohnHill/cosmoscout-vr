#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////
//                               This file is part of CosmoScout VR                               //
////////////////////////////////////////////////////////////////////////////////////////////////////

// SPDX-FileCopyrightText: German Aerospace Center (DLR) <cosmoscout@dlr.de>
// SPDX-License-Identifier: MIT

#include "gpuErrCheck.hpp"
#include "math.cuh"
#include "tiff_utils.hpp"
#include "with_atmosphere.cuh"

#include <cstdint>
#include <iostream>

////////////////////////////////////////////////////////////////////////////////////////////////////

struct Constants {
  double TOP_RADIUS;
  double BOTTOM_RADIUS;
  int    TRANSMITTANCE_TEXTURE_WIDTH;
  int    TRANSMITTANCE_TEXTURE_HEIGHT;
  int    SCATTERING_TEXTURE_MU_SIZE;
  int    SCATTERING_TEXTURE_MU_S_SIZE;
  int    SCATTERING_TEXTURE_NU_SIZE;
  double MU_S_MIN;
};

////////////////////////////////////////////////////////////////////////////////////////////////////

hipTextureObject_t createCudaTexture(tiff_utils::RGBATexture const& texture) {
  hipArray* cuArray;
  auto       channelDesc = hipCreateChannelDesc<float4>();

  hipMallocArray(&cuArray, &channelDesc, texture.width, texture.height);
  hipMemcpy2DToArray(cuArray, 0, 0, texture.data.data(), texture.width * sizeof(float) * 4,
      texture.width * sizeof(float) * 4, texture.height, hipMemcpyHostToDevice);

  // Specify texture object parameters
  hipResourceDesc resDesc = {};
  resDesc.resType          = hipResourceTypeArray;
  resDesc.res.array.array  = cuArray;

  hipTextureDesc texDesc  = {};
  texDesc.addressMode[0]   = hipAddressModeClamp;
  texDesc.addressMode[1]   = hipAddressModeClamp;
  texDesc.filterMode       = hipFilterModeLinear;
  texDesc.readMode         = hipReadModeElementType;
  texDesc.normalizedCoords = 1;

  // Create texture object
  hipTextureObject_t textureObject = 0;
  hipCreateTextureObject(&textureObject, &resDesc, &texDesc, nullptr);

  gpuErrchk(hipGetLastError());

  return textureObject;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ glm::vec2 intersectSphere(glm::dvec3 rayOrigin, glm::dvec3 rayDir, double radius) {
  double b   = glm::dot(rayOrigin, rayDir);
  double c   = glm::dot(rayOrigin, rayOrigin) - radius * radius;
  double det = b * b - c;

  if (det < 0.0) {
    return glm::dvec2(1, -1);
  }

  det = glm::sqrt(det);
  return glm::vec2(-b - det, -b + det);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

// Using acos is not very stable for small angles. This function is used to compute the angle
// between two vectors in a more stable way.
__device__ double angleBetweenVectors(glm::dvec3 u, glm::dvec3 v) {
  return 2.0 * glm::asin(0.5 * glm::length(u - v));
}

////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ glm::vec4 texture2D(hipTextureObject_t tex, glm::vec2 uv) {
  auto data = tex2D<float4>(tex, uv.x, uv.y);
  return glm::vec4(data.x, data.y, data.z, data.w);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ float safeSqrt(float a) {
  return glm::sqrt(glm::max(a, 0.0f));
}

////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ float clampDistance(float d) {
  return glm::max(d, 0.0f);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ float getTextureCoordFromUnitRange(float x, int textureSize) {
  return 0.5 / float(textureSize) + x * (1.0 - 1.0 / float(textureSize));
}

////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ float distanceToTopAtmosphereBoundary(Constants const& constants, float r, float mu) {
  float discriminant = r * r * (mu * mu - 1.0) + constants.TOP_RADIUS * constants.TOP_RADIUS;
  return clampDistance(-r * mu + safeSqrt(discriminant));
}

////////////////////////////////////////////////////////////////////////////////////////////////////

// As we are always in outer space, this function does not need the r parameter.
__device__ glm::vec2 getTransmittanceTextureUvFromRMu(Constants const& constants, double mu) {
  // Distance to top atmosphere boundary for a horizontal ray at ground level.
  double H = sqrt(constants.TOP_RADIUS * constants.TOP_RADIUS -
                  constants.BOTTOM_RADIUS * constants.BOTTOM_RADIUS);
  // Distance to the top atmosphere boundary for the ray (r,mu), and its minimum
  // and maximum values over all mu - obtained for (r,1) and (r,mu_horizon).
  double d    = distanceToTopAtmosphereBoundary(constants, constants.TOP_RADIUS, mu);
  double dMax = 2.0 * H;
  double xMu  = d / dMax;
  return glm::vec2(getTextureCoordFromUnitRange(xMu, constants.TRANSMITTANCE_TEXTURE_WIDTH),
      getTextureCoordFromUnitRange(1.0, constants.TRANSMITTANCE_TEXTURE_HEIGHT));
}

////////////////////////////////////////////////////////////////////////////////////////////////////

// As we are always in outer space, this function does not need the r parameter.
__device__ glm::vec3 getScatteringTextureUvwFromRMuMuSNu(
    Constants const& constants, double mu, double muS, double nu, bool rayRMuIntersectsGround) {

  // Distance to top atmosphere boundary for a horizontal ray at ground level.
  double H = sqrt(constants.TOP_RADIUS * constants.TOP_RADIUS -
                  constants.BOTTOM_RADIUS * constants.BOTTOM_RADIUS);

  // Discriminant of the quadratic equation for the intersections of the ray (r,mu) with the ground
  // (see rayIntersectsGround).
  double rMu          = constants.TOP_RADIUS * mu;
  double discriminant = rMu * rMu - constants.TOP_RADIUS * constants.TOP_RADIUS +
                        constants.BOTTOM_RADIUS * constants.BOTTOM_RADIUS;
  double uMu;
  if (rayRMuIntersectsGround) {
    // Distance to the ground for the ray (r,mu), and its minimum and maximum values over all mu -
    // obtained for (r,-1) and (r,mu_horizon).
    double d    = -rMu - safeSqrt(discriminant);
    double dMin = constants.TOP_RADIUS - constants.BOTTOM_RADIUS;
    double dMax = H;
    uMu = 0.5 - 0.5 * getTextureCoordFromUnitRange(dMax == dMin ? 0.0 : (d - dMin) / (dMax - dMin),
                          constants.SCATTERING_TEXTURE_MU_SIZE / 2);
  } else {
    // Distance to the top atmosphere boundary for the ray (r,mu), and its minimum and maximum
    // values over all mu - obtained for (r,1) and (r,mu_horizon).
    double d    = -rMu + safeSqrt(discriminant + H * H);
    double dMax = 2.0 * H;
    uMu         = 0.5 +
          0.5 * getTextureCoordFromUnitRange(d / dMax, constants.SCATTERING_TEXTURE_MU_SIZE / 2);
  }

  double d    = distanceToTopAtmosphereBoundary(constants, constants.BOTTOM_RADIUS, muS);
  double dMin = constants.TOP_RADIUS - constants.BOTTOM_RADIUS;
  double dMax = H;
  double a    = (d - dMin) / (dMax - dMin);
  double D =
      distanceToTopAtmosphereBoundary(constants, constants.BOTTOM_RADIUS, constants.MU_S_MIN);
  double A = (D - dMin) / (dMax - dMin);
  // An ad-hoc function equal to 0 for muS = MU_S_MIN (because then d = D and thus a = A), equal to
  // 1 for muS = 1 (because then d = dMin and thus a = 0), and with a large slope around muS = 0,
  // to get more texture samples near the horizon.
  float uMuS = getTextureCoordFromUnitRange(
      glm::max(1.0 - a / A, 0.0) / (1.0 + a), constants.SCATTERING_TEXTURE_MU_S_SIZE);

  float uNu = (nu + 1.0) / 2.0;
  return glm::vec3(uNu, uMuS, uMu);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ glm::vec3 getTransmittanceToTopAtmosphereBoundary(
    Constants const& constants, hipTextureObject_t transmittanceTexture, double mu) {
  glm::vec2 uv = getTransmittanceTextureUvFromRMu(constants, mu);
  return glm::vec3(texture2D(transmittanceTexture, uv));
}

////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ bool rayIntersectsGround(Constants const& constants, double mu) {
  return mu < 0.0 && constants.TOP_RADIUS * constants.TOP_RADIUS * (mu * mu - 1.0) +
                             constants.BOTTOM_RADIUS * constants.BOTTOM_RADIUS >=
                         0.0;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ glm::vec3 moleculePhaseFunction(hipTextureObject_t phaseTexture, float nu) {
  float theta = glm::acos(nu) / M_PI; // 0<->1
  return glm::vec3(texture2D(phaseTexture, glm::vec2(theta, 0.0)));
}

////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ glm::vec3 aerosolPhaseFunction(hipTextureObject_t phaseTexture, float nu) {
  float theta = glm::acos(nu) / M_PI; // 0<->1
  return glm::vec3(texture2D(phaseTexture, glm::vec2(theta, 1.0)));
}

////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ void getCombinedScattering(Constants const& constants,
    hipTextureObject_t                                multipleScatteringTexture,
    hipTextureObject_t singleAerosolsScatteringTexture, float mu, float muS, float nu,
    bool rayRMuIntersectsGround, glm::vec3& multipleScattering,
    glm::vec3& singleAerosolsScattering) {
  glm::vec3 uvw =
      getScatteringTextureUvwFromRMuMuSNu(constants, mu, muS, nu, rayRMuIntersectsGround);
  float     texCoordX = uvw.x * float(constants.SCATTERING_TEXTURE_NU_SIZE - 1);
  float     texX      = floor(texCoordX);
  float     lerp      = texCoordX - texX;
  glm::vec2 uv0 = glm::vec2((texX + uvw.y) / float(constants.SCATTERING_TEXTURE_NU_SIZE), uvw.z);
  glm::vec2 uv1 =
      glm::vec2((texX + 1.0 + uvw.y) / float(constants.SCATTERING_TEXTURE_NU_SIZE), uvw.z);

  multipleScattering = glm::vec3(texture2D(multipleScatteringTexture, uv0) * (1.0f - lerp) +
                                 texture2D(multipleScatteringTexture, uv1) * lerp);
  singleAerosolsScattering =
      glm::vec3(texture2D(singleAerosolsScatteringTexture, uv0) * (1.0f - lerp) +
                texture2D(singleAerosolsScatteringTexture, uv1) * lerp);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ glm::vec3 getSkyRadiance(Constants const& constants, hipTextureObject_t phaseTexture,
    hipTextureObject_t transmittanceTexture, hipTextureObject_t multipleScatteringTexture,
    hipTextureObject_t singleAerosolsScatteringTexture, glm::dvec3 camera, glm::dvec3 viewRay,
    glm::dvec3 sunDirection, glm::vec3& transmittance) {
  // Compute the distance to the top atmosphere boundary along the view ray, assuming the viewer is
  // in space (or NaN if the view ray does not intersect the atmosphere).
  double r   = length(camera);
  double rmu = dot(camera, viewRay);
  double distanceToTopAtmosphereBoundary =
      -rmu - sqrt(rmu * rmu - r * r + constants.TOP_RADIUS * constants.TOP_RADIUS);

  // If the view ray does not intersect the atmosphere, simply return 0.
  if (distanceToTopAtmosphereBoundary <= 0.0 && r > constants.TOP_RADIUS) {
    transmittance = glm::vec3(1.0);
    return glm::vec3(0.0);
  }

  camera += viewRay * distanceToTopAtmosphereBoundary;

  // Compute the mu, muS and nu parameters needed for the texture lookups.
  double mu                     = (rmu + distanceToTopAtmosphereBoundary) / constants.TOP_RADIUS;
  double muS                    = dot(camera, sunDirection) / constants.TOP_RADIUS;
  double nu                     = dot(viewRay, sunDirection);
  bool   rayRMuIntersectsGround = rayIntersectsGround(constants, mu);

  transmittance = rayRMuIntersectsGround ? glm::vec3(0.0)
                                         : getTransmittanceToTopAtmosphereBoundary(
                                               constants, transmittanceTexture, mu);

  glm::vec3 multipleScattering;
  glm::vec3 singleAerosolsScattering;
  getCombinedScattering(constants, multipleScatteringTexture, singleAerosolsScatteringTexture, mu,
      muS, nu, rayRMuIntersectsGround, multipleScattering, singleAerosolsScattering);

  return multipleScattering * moleculePhaseFunction(phaseTexture, nu) +
         singleAerosolsScattering * aerosolPhaseFunction(phaseTexture, nu);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void drawPlanet(float* shadowMap, ShadowSettings settings, LimbDarkening limbDarkening,
    Constants constants, hipTextureObject_t multiscatteringTexture,
    hipTextureObject_t singleScatteringTexture, hipTextureObject_t thetaDeviationTexture,
    hipTextureObject_t phaseTexture, hipTextureObject_t transmittanceTexture) {

  uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
  uint32_t i = y * settings.size + x;

  if ((x >= settings.size) || (y >= settings.size)) {
    return;
  }

  // Horizon close up from quite close to the Earth.
  // glm::dvec3 camera       = glm::dvec3(0.0, constants.BOTTOM_RADIUS, 1000000.0);
  // double     fieldOfView  = 0.05 * M_PI;
  // glm::dvec3 sunDirection = glm::normalize(glm::vec3(0.0, 0.0, -1.0));
  // float      exposure     = 0.001;

  // Horizon close up from Moon.
  glm::dvec3 camera       = glm::dvec3(0.0, constants.BOTTOM_RADIUS, 300000000.0);
  double     fieldOfView  = 0.005 * M_PI;
  glm::dvec3 sunDirection = glm::normalize(glm::vec3(0.0, 0.0, -1.0));
  float      exposure     = 0.0001;

  // Total eclipse from Moon.
  // glm::dvec3 camera       = glm::dvec3(0.0, 0.0, 300000000.0);
  // double     fieldOfView  = 0.018 * M_PI;
  // glm::dvec3 sunDirection = glm::normalize(glm::vec3(0.0, 0.0, -1.0));
  // float      exposure     = 0.001;

  // Compute the direction of the ray.
  double theta = (x / (double)settings.size - 0.5) * fieldOfView;
  double phi   = (y / (double)settings.size - 0.5) * fieldOfView;

  glm::dvec3 rayDir =
      glm::dvec3(glm::sin(theta) * glm::cos(phi), glm::sin(phi), -glm::cos(theta) * glm::cos(phi));
  glm::vec3 transmittance;

  glm::vec3 skyRadiance = getSkyRadiance(constants, phaseTexture, transmittanceTexture,
      multiscatteringTexture, singleScatteringTexture, camera, rayDir, sunDirection, transmittance);

  float sunAngularRadius = 0.0082 / 2.0;

  glm::vec3 sunRadiance =
      transmittance *
      glm::vec3(
          limbDarkening.get(angleBetweenVectors(rayDir, sunDirection) / sunAngularRadius) * 1.1e9);

  shadowMap[i * 3 + 0] = (skyRadiance.r + sunRadiance.r) * exposure;
  shadowMap[i * 3 + 1] = (skyRadiance.g + sunRadiance.g) * exposure;
  shadowMap[i * 3 + 2] = (skyRadiance.b + sunRadiance.b) * exposure;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void computeAtmosphereShadow(float* shadowMap, ShadowSettings settings,
    std::string const& atmosphereSettings, LimbDarkening limbDarkening) {
  // Compute the 2D kernel size.
  dim3     blockSize(16, 16);
  uint32_t numBlocksX = (settings.size + blockSize.x - 1) / blockSize.x;
  uint32_t numBlocksY = (settings.size + blockSize.y - 1) / blockSize.y;
  dim3     gridSize   = dim3(numBlocksX, numBlocksY);

  tiff_utils::RGBATexture multiscattering =
      tiff_utils::read2DTexture(atmosphereSettings + "/multiple_scattering.tif", 31);
  tiff_utils::RGBATexture singleScattering =
      tiff_utils::read2DTexture(atmosphereSettings + "/single_aerosols_scattering.tif", 31);
  tiff_utils::RGBATexture theta_deviation =
      tiff_utils::read2DTexture(atmosphereSettings + "/theta_deviation.tif");
  tiff_utils::RGBATexture phase = tiff_utils::read2DTexture(atmosphereSettings + "/phase.tif");
  tiff_utils::RGBATexture transmittance =
      tiff_utils::read2DTexture(atmosphereSettings + "/transmittance.tif");

  std::cout << "Computing shadow map with atmosphere..." << std::endl;
  std::cout << "  - Mutli-scattering texture dimensions: " << multiscattering.width << "x"
            << multiscattering.height << std::endl;
  std::cout << "  - Single-scattering texture dimensions: " << singleScattering.width << "x"
            << singleScattering.height << std::endl;
  std::cout << "  - Theta deviation texture dimensions: " << theta_deviation.width << "x"
            << theta_deviation.height << std::endl;
  std::cout << "  - Phase texture dimensions: " << phase.width << "x" << phase.height << std::endl;
  std::cout << "  - Transmittance texture dimensions: " << transmittance.width << "x"
            << transmittance.height << std::endl;

  hipTextureObject_t multiscatteringTexture  = createCudaTexture(multiscattering);
  hipTextureObject_t singleScatteringTexture = createCudaTexture(singleScattering);
  hipTextureObject_t thetaDeviationTexture   = createCudaTexture(theta_deviation);
  hipTextureObject_t phaseTexture            = createCudaTexture(phase);
  hipTextureObject_t transmittanceTexture    = createCudaTexture(transmittance);

  Constants constants;
  constants.BOTTOM_RADIUS                = 6371000.0;
  constants.TOP_RADIUS                   = 6371000.0 + 100000.0;
  constants.TRANSMITTANCE_TEXTURE_WIDTH  = 256;
  constants.TRANSMITTANCE_TEXTURE_HEIGHT = 64;
  constants.SCATTERING_TEXTURE_MU_SIZE   = 128;
  constants.SCATTERING_TEXTURE_MU_S_SIZE = 256 / 8;
  constants.SCATTERING_TEXTURE_NU_SIZE   = 8;
  constants.MU_S_MIN                     = std::cos(2.094395160675049);

  drawPlanet<<<gridSize, blockSize>>>(shadowMap, settings, limbDarkening, constants,
      multiscatteringTexture, singleScatteringTexture, thetaDeviationTexture, phaseTexture,
      transmittanceTexture);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
