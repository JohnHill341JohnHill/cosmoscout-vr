////////////////////////////////////////////////////////////////////////////////////////////////////
//                               This file is part of CosmoScout VR                               //
////////////////////////////////////////////////////////////////////////////////////////////////////

// SPDX-FileCopyrightText: German Aerospace Center (DLR) <cosmoscout@dlr.de>
// SPDX-License-Identifier: MIT

#include "../../src/cs-utils/CommandLine.hpp"

#include "LimbDarkening.cuh"
#include "without_atmosphere.cuh"
#include "math.cuh"
#include "types.hpp"

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include <stb_image.h>
#include <stb_image_write.h>

////////////////////////////////////////////////////////////////////////////////////////////////////
// This tool can be used to create the eclipse shadow maps used by CosmoScout VR. See the         //
// README.md file in this directory for usage instructions!                                       //
////////////////////////////////////////////////////////////////////////////////////////////////////

// This macro is used in multiple locations to check for Cuda errors.
// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define gpuErrchk(ans)                                                                             \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) {
      exit(code);
    }
  }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char** argv) {

  stbi_flip_vertically_on_write(1);

  ShadowSettings settings;

  std::string cOutput    = "shadow.hdr";
  std::string cMode      = "limb-darkening";
  bool        cPrintHelp = false;

  // First configure all possible command line options.
  cs::utils::CommandLine args(
      "Welcome to the shadow map generator! Here are the available options:");
  args.addArgument({"-o", "--output"}, &cOutput,
      "The image will be written to this file (default: \"" + cOutput + "\").");
  args.addArgument({"--size"}, &settings.size,
      "The output texture size (default: " + std::to_string(settings.size) + ").");
  args.addArgument({"--mode"}, &cMode,
      "This should be either 'limb-darkening', 'circles', 'linear', or 'smoothstep' (default: " +
          cMode + ").");
  args.addArgument({"--with-umbra"}, &settings.includeUmbra,
      "Add the umbra region to the shadow map (default: " + std::to_string(settings.includeUmbra) +
          ").");
  args.addArgument({"--mapping-exponent"}, &settings.mappingExponent,
      "Adjusts the distribution of sampling positions. A value of 1.0 will position the "
      "umbra's end in the middle of the texture, larger values will shift this to the "
      "right. (default: " +
          std::to_string(settings.mappingExponent) + ").");
  args.addArgument({"-h", "--help"}, &cPrintHelp, "Show this help message.");

  // Then do the actual parsing.
  try {
    std::vector<std::string> arguments(argv + 1, argv + argc);
    args.parse(arguments);
  } catch (std::runtime_error const& e) {
    std::cerr << "Failed to parse command line arguments: " << e.what() << std::endl;
    return 1;
  }

  // When cPrintHelp was set to true, we print a help message and exit.
  if (cPrintHelp) {
    args.printHelp();
    return 0;
  }

  // Check whether a valid mode was given.
  if (cMode != "limb-darkening" && cMode != "circles" && cMode != "linear" &&
      cMode != "smoothstep") {
    std::cerr << "Invalid value given for --mode!" << std::endl;

    return 1;
  }

  // Initialize the limb darkening model.
  LimbDarkening limbDarkening;
  limbDarkening.init();

  // Compute the 2D kernel size.
  dim3     blockSize(16, 16);
  uint32_t numBlocksX = (settings.size + blockSize.x - 1) / blockSize.x;
  uint32_t numBlocksY = (settings.size + blockSize.y - 1) / blockSize.y;
  dim3     gridSize   = dim3(numBlocksX, numBlocksY);

  // Allocate the shared memory for the shadow map.
  float* shadow = nullptr;
  gpuErrchk(hipMallocManaged(
      &shadow, static_cast<size_t>(settings.size * settings.size) * sizeof(float)));

  // Compute the shadow map based on the given mode.
  if (cMode == "limb-darkening") {
    computeLimbDarkeningShadow<<<gridSize, blockSize>>>(shadow, settings, limbDarkening);
  } else if (cMode == "circles") {
    computeCircleIntersectionShadow<<<gridSize, blockSize>>>(shadow, settings);
  } else if (cMode == "linear") {
    computeLinearShadow<<<gridSize, blockSize>>>(shadow, settings);
  } else if (cMode == "smoothstep") {
    computeSmoothstepShadow<<<gridSize, blockSize>>>(shadow, settings);
  }

  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  // Finally write the output texture!
  stbi_write_hdr(
      cOutput.c_str(), static_cast<int>(settings.size), static_cast<int>(settings.size), 1, shadow);

  return 0;
}
