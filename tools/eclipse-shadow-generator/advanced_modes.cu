#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////
//                               This file is part of CosmoScout VR                               //
////////////////////////////////////////////////////////////////////////////////////////////////////

// SPDX-FileCopyrightText: German Aerospace Center (DLR) <cosmoscout@dlr.de>
// SPDX-License-Identifier: MIT

#include "advanced_modes.cuh"

#include "atmosphere_rendering.cuh"
#include "common.hpp"
#include "gpuErrCheck.hpp"
#include "math.cuh"

#include <stb_image_write.h>

#include <cstdint>
#include <iostream>

////////////////////////////////////////////////////////////////////////////////////////////////////

namespace {

////////////////////////////////////////////////////////////////////////////////////////////////////

enum class Mode { eBruneton, ePlanetView, eAtmoView };

////////////////////////////////////////////////////////////////////////////////////////////////////

// Tonemapping code and color space conversions.
// http://filmicworlds.com/blog/filmic-tonemapping-operators/

__device__ glm::vec3 uncharted2Tonemap(glm::vec3 c) {
  const float A = 0.15;
  const float B = 0.50;
  const float C = 0.10;
  const float D = 0.20;
  const float E = 0.02;
  const float F = 0.30;
  return ((c * (A * c + C * B) + D * E) / (c * (A * c + B) + D * F)) - E / F;
}

__device__ glm::vec3 tonemap(glm::vec3 c) {
  const float W        = 11.2;
  c                    = uncharted2Tonemap(10.0f * c);
  glm::vec3 whiteScale = glm::vec3(1.0) / uncharted2Tonemap(glm::vec3(W));
  return c * whiteScale;
}

__device__ float linearToSRGB(float c) {
  if (c <= 0.0031308f)
    return 12.92f * c;
  else
    return 1.055f * pow(c, 1.0f / 2.4f) - 0.055f;
}

__device__ glm::vec3 linearToSRGB(glm::vec3 c) {
  return glm::vec3(linearToSRGB(c.r), linearToSRGB(c.g), linearToSRGB(c.b));
}

////////////////////////////////////////////////////////////////////////////////////////////////////

double __device__ getSunIlluminance(double sunDistance) {
  const double sunLuminousPower = 3.75e28;
  return sunLuminousPower / (4.0 * glm::pi<double>() * sunDistance * sunDistance);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void computeShadowMap(common::OutputSettings output, common::MappingSettings mapping,
    common::GeometrySettings geometry, common::LimbDarkening limbDarkening,
    advanced::Textures textures) {

  uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
  uint32_t i = y * output.mSize + x;

  if ((x >= output.mSize) || (y >= output.mSize)) {
    return;
  }

  uint32_t  samplesX = 64;
  uint32_t  samplesY = 64;
  glm::vec3 indirectIlluminance(0.0);

  double   phiOcc, phiSun, delta;
  uint32_t iterations = math::mapPixelToAngles(
      glm::ivec2(x, y), output.mSize, mapping, geometry, phiOcc, phiSun, delta);

  if (i == 100) {
    printf("phiSun: %f\n", phiSun);
    printf("phiOcc: %f\n", phiOcc);
    printf("delta: %f\n", delta);
    printf("iterations: %d\n", iterations);
  }

  double occDist    = geometry.mRadiusOcc / glm::sin(phiOcc);
  double sunDist    = geometry.mRadiusSun / glm::sin(phiSun);
  double atmoRadius = geometry.mRadiusAtmo;
  double phiAtmo    = glm::asin(atmoRadius / occDist);

  glm::dvec3 camera       = glm::dvec3(0.0, 0.0, occDist);
  glm::dvec3 sunDirection = glm::dvec3(0.0, glm::sin(delta), -glm::cos(delta));

  // Compute the direction of the ray.

  for (uint32_t sampleY = 0; sampleY < samplesY; ++sampleY) {
    double altitude         = ((double)sampleY + 0.5) / samplesY;
    double upperAltitude    = ((double)sampleY + 1.0) / samplesY;
    double lowerAltitude    = ((double)sampleY) / samplesY;
    double upperPhiRay      = phiOcc + upperAltitude * (phiAtmo - phiOcc);
    double lowerPhiRay      = phiOcc + lowerAltitude * (phiAtmo - phiOcc);
    double rowSolidAngle    = 0.5 * (math::getCapArea(upperPhiRay) - math::getCapArea(lowerPhiRay));
    double sampleSolidAngle = rowSolidAngle / samplesX;

    for (uint32_t sampleX = 0; sampleX < samplesX; ++sampleX) {

      double theta = (((double)sampleX + 0.5) / samplesX) * M_PI;

      double     phiRay = phiOcc + altitude * (phiAtmo - phiOcc);
      glm::dvec3 rayDir = glm::dvec3(0.0, glm::sin(phiRay), -glm::cos(phiRay));
      rayDir =
          glm::normalize(math::rotateVector(rayDir, glm::dvec3(0.0, 0.0, -1.0), glm::cos(theta)));

      glm::vec3 luminance = advanced::getLuminance(
          camera, rayDir, sunDirection, geometry, limbDarkening, textures, phiSun);

      indirectIlluminance += luminance * (float)(sampleSolidAngle / getSunIlluminance(sunDist));
    }
  }

  double sunArea = math::getCircleArea(1.0);
  double radiusOcc, distance;
  math::mapPixelToRadii(glm::ivec2(x, y), output.mSize, mapping, radiusOcc, distance);
  double directIlluminance =
      1.0 - math::sampleCircleIntersection(1.0,
                radiusOcc * geometry.mRadiusAtmo / geometry.mRadiusOcc, distance, limbDarkening) /
                sunArea;

  output.mBuffer[i * 3 + 0] = indirectIlluminance.r + directIlluminance;
  output.mBuffer[i * 3 + 1] = indirectIlluminance.g + directIlluminance;
  output.mBuffer[i * 3 + 2] = indirectIlluminance.b + directIlluminance;
}
////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void drawAtmoView(common::MappingSettings mapping, common::GeometrySettings geometry,
    float exposure, double phiOcc, double phiSun, double delta, common::OutputSettings output,
    common::LimbDarkening limbDarkening, advanced::Textures textures) {

  uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
  uint32_t i = y * output.mSize + x;

  if ((x >= output.mSize) || (y >= output.mSize)) {
    return;
  }

  double occDist    = geometry.mRadiusOcc / glm::sin(phiOcc);
  double atmoRadius = geometry.mRadiusAtmo;
  double phiAtmo    = glm::asin(atmoRadius / occDist);

  glm::dvec3 camera       = glm::dvec3(0.0, 0.0, occDist);
  glm::dvec3 sunDirection = glm::dvec3(0.0, glm::sin(delta), -glm::cos(delta));

  // Compute the direction of the ray.
  double theta    = (x / (double)output.mSize) * M_PI;
  double altitude = (y / (double)output.mSize);

  double     phiRay = phiOcc + altitude * (phiAtmo - phiOcc);
  glm::dvec3 rayDir = glm::dvec3(0.0, glm::sin(phiRay), -glm::cos(phiRay));
  rayDir = glm::normalize(math::rotateVector(rayDir, glm::dvec3(0.0, 0.0, -1.0), glm::cos(theta)));

  glm::vec3 luminance = advanced::getLuminance(
      camera, rayDir, sunDirection, geometry, limbDarkening, textures, phiSun);

  luminance = linearToSRGB(tonemap(luminance * exposure));

  output.mBuffer[i * 3 + 0] = luminance.r;
  output.mBuffer[i * 3 + 1] = luminance.g;
  output.mBuffer[i * 3 + 2] = luminance.b;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void drawPlanet(common::MappingSettings mapping, common::GeometrySettings geometry,
    float exposure, double phiOcc, double phiSun, double delta, float fov,
    common::OutputSettings output, common::LimbDarkening limbDarkening,
    advanced::Textures textures) {

  uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
  uint32_t i = y * output.mSize + x;

  if ((x >= output.mSize) || (y >= output.mSize)) {
    return;
  }

  // Total eclipse from Moon, horizon close up.
  double     occDist      = geometry.mRadiusOcc / glm::sin(phiOcc);
  glm::dvec3 camera       = glm::dvec3(0.0, 0.0, occDist);
  double     fieldOfView  = fov * M_PI / 180.0;
  glm::dvec3 sunDirection = glm::dvec3(0.0, glm::sin(delta), -glm::cos(delta));

  // Compute the direction of the ray.
  double theta = (x / (double)output.mSize - 0.5) * fieldOfView;
  double phi   = (y / (double)output.mSize - 0.5) * fieldOfView;

  glm::dvec3 rayDir =
      glm::dvec3(glm::sin(theta) * glm::cos(phi), glm::sin(phi), -glm::cos(theta) * glm::cos(phi));

  glm::vec3 luminance = advanced::getLuminance(
      camera, rayDir, sunDirection, geometry, limbDarkening, textures, phiSun);

  luminance = linearToSRGB(tonemap(luminance * exposure));

  output.mBuffer[i * 3 + 0] = luminance.r;
  output.mBuffer[i * 3 + 1] = luminance.g;
  output.mBuffer[i * 3 + 2] = luminance.b;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

int run(Mode mode, std::vector<std::string> const& arguments) {

  std::string              input;
  common::MappingSettings  mapping;
  common::OutputSettings   output;
  common::GeometrySettings geometry;
  bool                     printHelp = false;

  // These are only required for the planet or atmosphere view modes.
  float exposure = 0.0001; // The exposure of the image used during tonemapping.
  float x        = 0.5;    // The shadow map x coordinate for which to render the view.
  float y        = 0.5;    // The shadow map y coordinate for which to render the view.

  // This is only required for the planet view mode.
  float fov = 45.0; // The field of view of the camera in degrees.

  // First configure all possible command line options.
  cs::utils::CommandLine args("Here are the available options:");
  common::addMappingSettingsFlags(args, mapping);
  common::addOutputSettingsFlags(args, output);
  common::addGeometrySettingsFlags(args, geometry);

  args.addArgument({"--input"}, &input, "The path to the atmosphere settings directory.");

  if (mode == Mode::eAtmoView || mode == Mode::ePlanetView) {
    args.addArgument({"--exposure"}, &exposure,
        "The exposure of the image. Default is " + std::to_string(exposure));
    args.addArgument({"--x"}, &x,
        "The shadow map x coordinate for which to render the view. "
        "Default is " +
            std::to_string(x));
    args.addArgument({"--y"}, &y,
        "The shadow map y coordinate for which to render the view. "
        "Default is " +
            std::to_string(y));
  }

  if (mode == Mode::ePlanetView) {
    args.addArgument({"--fov"}, &fov,
        "The field of view of the camera in degrees. Default is " + std::to_string(fov));
  }

  args.addArgument({"-h", "--help"}, &printHelp, "Show this help message.");

  // Then do the actual parsing.
  try {
    args.parse(arguments);
  } catch (std::runtime_error const& e) {
    std::cerr << "Failed to parse command line arguments: " << e.what() << std::endl;
    return 1;
  }

  // When printHelp was set to true, we print a help message and exit.
  if (printHelp) {
    args.printHelp();
    return 0;
  }

  // If we are in atmosphere mode, we need also the atmosphere settings.
  if (input.empty()) {
    std::cerr << "When using the 'bruneton', 'planet-view', or 'atmo-view' mode, you must provide "
                 "the path to the atmosphere settings directory using --input!"
              << std::endl;
    return 1;
  }

  // Load the atmosphere settings.
  auto textures = advanced::loadTextures(input);

  // Initialize the limb darkening model.
  common::LimbDarkening limbDarkening;
  limbDarkening.init();

  // Compute the 2D kernel size.
  dim3     blockSize(16, 16);
  uint32_t numBlocksX = (output.mSize + blockSize.x - 1) / blockSize.x;
  uint32_t numBlocksY = (output.mSize + blockSize.y - 1) / blockSize.y;
  dim3     gridSize   = dim3(numBlocksX, numBlocksY);

  // Allocate the shared memory for the shadow map.
  gpuErrchk(hipMallocManaged(
      &output.mBuffer, static_cast<size_t>(output.mSize * output.mSize) * 3 * sizeof(float)));

  if (mode == Mode::eBruneton) {
    computeShadowMap<<<gridSize, blockSize>>>(output, mapping, geometry, limbDarkening, textures);
  } else {

    double   phiOcc, phiSun, delta;
    uint32_t iterations = math::mapPixelToAngles(glm::ivec2(x * output.mSize, y * output.mSize),
        output.mSize, mapping, geometry, phiOcc, phiSun, delta);

    std::cout << "Required " << iterations << " iterations to find the correct angles."
              << std::endl;

    if (mode == Mode::ePlanetView) {
      drawPlanet<<<gridSize, blockSize>>>(
          mapping, geometry, exposure, phiOcc, phiSun, delta, fov, output, limbDarkening, textures);
    } else if (mode == Mode::eAtmoView) {
      drawAtmoView<<<gridSize, blockSize>>>(
          mapping, geometry, exposure, phiOcc, phiSun, delta, output, limbDarkening, textures);
    }
  }

  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  // Finally write the output texture!
  stbi_write_hdr(output.mFile.c_str(), static_cast<int>(output.mSize),
      static_cast<int>(output.mSize), 3, output.mBuffer);

  // Free the shared memory.
  gpuErrchk(hipFree(output.mBuffer));

  return 0;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace

////////////////////////////////////////////////////////////////////////////////////////////////////

namespace advanced {

////////////////////////////////////////////////////////////////////////////////////////////////////

int brunetonMode(std::vector<std::string> const& arguments) {
  return run(Mode::eBruneton, arguments);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

int planetViewMode(std::vector<std::string> const& arguments) {
  return run(Mode::ePlanetView, arguments);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

int atmoViewMode(std::vector<std::string> const& arguments) {
  return run(Mode::eAtmoView, arguments);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace advanced

////////////////////////////////////////////////////////////////////////////////////////////////////
